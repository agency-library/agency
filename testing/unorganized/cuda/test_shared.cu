#include "hip/hip_runtime.h"
#include <agency/agency.hpp>
#include <agency/cuda.hpp>
#include <thrust/device_vector.h>
#include <cstdio>
#include <cassert>


template<class T>
__device__
T fetch_and_add(T* ptr, T value)
{
#ifdef __NVCC__
  return atomicAdd(ptr, value);
#else
  return __sync_fetch_and_add(ptr, value);
#endif
}


using cuda_thread = agency::parallel_group<agency::concurrent_agent>;


struct functor
{
  __device__
  void operator()(cuda_thread& self, int* outer_result, int& outer_shared, int& inner_shared)
  {
    printf("idx: {%zu, %zu}\n", self.outer().index(), self.inner().index());
    printf("outer_shared: %d\n", outer_shared);
    printf("inner_shared: %d\n", inner_shared);

    fetch_and_add(&inner_shared, 1);
    self.inner().wait();

#if (defined __APPLE__  || defined __MACOSX)
    // assert is not supported on OSX, use printf if result is incorrect
    if(!(inner_shared == self.inner().group_size() + 2))
    {
      printf(" -- failure -- : return\n");
      return;
    }
#else
    assert(inner_shared == self.inner().group_size() + 2);
#endif

    auto result = fetch_and_add(&outer_shared, 1);

    // exactly one agent will see this result
    if(result == (2 * 2))
    {
      *outer_result = result + 1;
    }
  }
};


int main()
{
  using cuda_thread = agency::parallel_group<agency::concurrent_agent>;

  auto policy = agency::cuda::par(2, agency::cuda::con(2));

  thrust::device_vector<int> outer_result(1);

  agency::bulk_invoke(policy, functor(), thrust::raw_pointer_cast(outer_result.data()), agency::share_at_scope<0>(1), agency::share_at_scope<1>(2));

  assert(outer_result[0] == (2 * 2 + 1));

  std::cout << "OK" << std::endl;

  return 0;
}

